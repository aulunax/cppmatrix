#include "hip/hip_runtime.h"
#include "cudaOps.h"
#include <>


__global__ void matrixMul(double* a, double* b, double* c, int rowsA, int colsA, int colsB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rowsA && col < colsB) {
        double sum = 0.0;
        for (int k = 0; k < colsA; ++k) {
            sum += a[row * colsA + k] * b[k * colsB + col];
        }
        c[row * colsB + col] = sum;
    }
}

// Host function to perform matrix multiplication
void mulMatrix(const Matrix<double>& a, const Matrix<double>& b, Matrix<double>& c) {
    int rowsA = a.getSize().n;
    int colsA = a.getSize().m;
    int colsB = b.getSize().m;

    // Allocate device memory
    double* dev_a, *dev_b, *dev_c;
    hipMalloc((void**)&dev_a, rowsA * colsA * sizeof(double));
    hipMalloc((void**)&dev_b, colsA * colsB * sizeof(double));
    hipMalloc((void**)&dev_c, rowsA * colsB * sizeof(double));

    // Copy input matrices from host to device
    hipMemcpy(dev_a, a.getDataVec().data(), rowsA * colsA * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b.getDataVec().data(), colsA * colsB * sizeof(double), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((colsB + blockDim.x - 1) / blockDim.x, (rowsA + blockDim.y - 1) / blockDim.y);

    // Launch kernel
    matrixMul<<<gridDim, blockDim>>>(dev_a, dev_b, dev_c, rowsA, colsA, colsB);

    // Copy result matrix from device to host
    hipMemcpy(c.getDataVec().data(), dev_c, rowsA * colsB * sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}